#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "float.h"
//#include "hip/hip_runtime_api.h"
#include "volumehelper.h"
//#include "volumehelper.cuh"


#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int  uint;
typedef unsigned char uchar;
//hipArray *d_volumeArray = 0;
hipArray *d_volumeArray;
hipArray *d_transferFuncArray;
int size = 0;
int3 dim = { 0,0,0 };


texture<uchar, hipTextureType3D, hipReadModeNormalizedFloat> volumeTex;
texture<float4, hipTextureType3D, hipReadModeElementType> transferTex;
surface<void, cudaSurfaceType3D> volumeSurf;
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

#include "raycaster.cuh"

__global__ void logScaleData(uchar* pIinput, int withLog, int3 pDim) {
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;
	if (x < pDim.x && y < pDim.y && z < pDim.z) {
		float max = 255;
		uchar dens = pIinput[x + y * pDim.x + z * pDim.x*pDim.y];
		if (withLog)
		{
			float result = (float)dens;
			float maxLog = log(max + 1);
			result = round(255 * (log(result + 1) / maxLog));
			dens = (uchar)result;
		}
		surf3Dwrite(dens, volumeSurf, x * sizeof(uchar), y, z, hipBoundaryModeClamp);
	}
}



__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
__global__ void blur( float* source, float *output,  int3 pDim, int r)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;
	int l = 2 * r + 1;
	if (x < pDim.x && y < pDim.y && z < pDim.z)
	{
		float kerneltotal = 0;
		int i = 0;
		if (source[x + pDim.x * y + pDim.x * pDim.y*z] != 0)
		{
			for (int xi = x - r, xii = 0; xi <= x + r; xi++, xii++)
			{
				for (int yi = y - r, yii = 0; yi <= y + r; yi++, yii++)
				{
					for (int zi = z - r, zii = 0; zi <= z + r; zi++, zii++)
					{
						if (xi >= 0 && xi < pDim.x && yi >= 0 && yi < pDim.y && zi >= 0 && zi < pDim.z)
						{
							//float source = 
							output[x + pDim.x * y + pDim.x * pDim.y*z] += (source[xi + pDim.x*yi  + pDim.x*pDim.y*zi])/((2*r+1)*(2*r+1)*(2*r+1));
							//output[x + dimx * y + dimx * dimy*z] += kernel[i] * source[xi + dimx * yi + dimx * dimy*zi];
							//kerneltotal += kernel[i];

						}
						i++;
					}
				}
			}
			if (kerneltotal != 0)
			{
				//  output[x + dimx*y + dimx*dimy*z] /= kerneltotal;
			}
		}
	}
}
int iDivUp(int a, int b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

extern "C" void logScale( std::vector<unsigned char> pInput,  bool pWithLog ) {
	unsigned char *dev_input = 0;
	int size = dim.x * dim.y*dim.z;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_input, size * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_input, pInput.data(), size * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 blockSize(8, 8, 1);
	dim3 gridSize(iDivUp(dim.x, blockSize.x), iDivUp(dim.y, blockSize.y), iDivUp(dim.z, blockSize.z));

	// Launch a kernel on the GPU with one thread for each element.
	logScaleData <<< gridSize, blockSize >>> (dev_input, pWithLog ?1:0, dim);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

Error:
	hipFree(dev_input);

}

extern "C" void blurData(std::vector<float> pInput, std::vector<float>& pOutput, int3 pDim, int pRadius) {
	float *dev_input = 0;
	float *dev_output = 0;
	int size = pDim.x * pDim.y*pDim.z;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_input, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_output, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_input, pInput.data(), size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 blockSize(8,8,1);
	dim3 gridSize(iDivUp(pDim.x, blockSize.x), iDivUp(pDim.y, blockSize.y), iDivUp(pDim.z, blockSize.z));

	// Launch a kernel on the GPU with one thread for each element.
	blur <<< gridSize, blockSize >>> ( dev_input, dev_output, pDim, pRadius);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(pOutput.data(), dev_output, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_output);
	hipFree(dev_input);
	
}

extern "C" void initCuda( std::vector<unsigned char> h_volume, int3 pDim)
{
	size = pDim.x *pDim.y*pDim.z;
	dim = pDim;
	
	//create 3D global texture
	hipChannelFormatDesc channelDescVolume = hipCreateChannelDesc<unsigned char>();
	hipExtent vol_dim = { pDim.x, pDim.y, pDim.z };
	hipMalloc3DArray(&d_volumeArray, &channelDescVolume, vol_dim, hipArraySurfaceLoadStore);
	// copy data to 3D array
	hipMemcpy3DParms copyParamsVol = { 0 };
	copyParamsVol.srcPtr = make_hipPitchedPtr(h_volume.data(), vol_dim.width * sizeof(unsigned char), vol_dim.width, vol_dim.height);
	copyParamsVol.dstArray = d_volumeArray;
	copyParamsVol.extent = vol_dim;
	copyParamsVol.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParamsVol);

	// set texture parameters
	volumeTex.normalized = true;                      // access with normalized texture coordinates
	volumeTex.filterMode = hipFilterModeLinear;      // linear interpolation
	volumeTex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	volumeTex.addressMode[1] = hipAddressModeClamp;
	//volumeTex.addressMode[2] = hipAddressModeClamp;

	// Bind the array to the texture
	hipBindTextureToArray(volumeTex, d_volumeArray, channelDescVolume);
	checkCudaErrors(cudaBindSurfaceToArray(volumeSurf, d_volumeArray));
	//---------------------transfer tex---------------------------------------------------------

	// create transfer function texture
	float4 transferFunc[] =
	{
		//----------		
			{ 1.0, 0.0, 0.0, 0.0, },
			{ 1.0, 0.0, 0.0, 1.0, },
			{ 1.0, 0.5, 0.0, 1.0, },
			{ 1.0, 1.0, 0.0, 1.0, },
			{ 0.0, 1.0, 0.0, 1.0, },
			{ 0.0, 1.0, 1.0, 1.0, },
			{ 0.0, 0.0, 1.0, 1.0, },
			{ 0.0, 0.0, 1.0, 1.0, },
			{ 1.0, 0.0, 0.0, 1.0, },
		//----------		
			{ 1.0, 0.0, 0.0, 0.0, },
			{ 1.0, 0.0, 0.0, 0.0, },
			{ 1.0, 0.5, 0.0, 0.0, },
			{ 1.0, 1.0, 0.0, 1.0, },
			{ 0.0, 1.0, 0.0, 1.0, },
			{ 0.0, 1.0, 1.0, 1.0, },
			{ 0.0, 0.0, 1.0, 1.0, },
			{ 0.0, 0.0, 1.0, 1.0, },
			{ 1.0, 0.0, 0.0, 1.0, },
		//----------		
			{ 1.0, 0.0, 0.0, 0.0, },
			{ 1.0, 0.0, 0.0, 0.0, },
			{ 1.0, 0.5, 0.0, 0.0, },
			{ 1.0, 1.0, 0.0, 0.0, },
			{ 0.0, 1.0, 0.0, 0.0, },
			{ 0.0, 1.0, 1.0, 0.5, },
			{ 0.0, 0.0, 1.0, 1.0, },
			{ 0.0, 0.0, 1.0, 1.0, },
			{ 1.0, 0.0, 0.0, 1.0, },
	};


	//hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
	//checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc) / sizeof(float4), 1));
	//checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

	//transferTex.filterMode = hipFilterModeLinear;
	//transferTex.normalized = true;    // access with normalized texture coordinates
	//transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

	//													 // Bind the array to the texture
	//checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
	hipExtent tf_dim = { 9, 3, 1 };
	hipMalloc3DArray(&d_transferFuncArray, &channelDesc, tf_dim);
	// copy data to 3D array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(transferFunc, tf_dim.width * sizeof(float4), tf_dim.width, tf_dim.height);
	copyParams.dstArray = d_transferFuncArray;
	copyParams.extent = tf_dim;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	
	// set texture parameters
	transferTex.normalized = true;                      // access with normalized texture coordinates
	transferTex.filterMode = hipFilterModeLinear;      // linear interpolation
	transferTex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	transferTex.addressMode[1] = hipAddressModeClamp;

											  //Bind the array to the texture
	hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc);
}
//
//extern "C" void updateTF(unsigned char * colors, float index, dim3 blockSize, dim3 gridSize, float opacity, float* debug)
//{
//	float* dev_debug;
//	hipMalloc((void**)&dev_debug, 5 * sizeof(float));
//	hipMemcpy(dev_debug, debug, 5 * sizeof(float), hipMemcpyHostToDevice);
//	updateColors<<<gridSize, blockSize>>>(colors, dev_volume, index, dimx, dimy, dimz, opacity,dev_debug);
//	// Check for any errors launching the kernel
//	hipError_t cudaStatus = hipGetLastError();
//	const char * msg = hipGetErrorString(cudaStatus);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//	}
//	hipMemcpy(debug, dev_debug, 5 * sizeof(float), hipMemcpyDeviceToHost);
//	//debug[0] = 10;
//}

//
//
//extern"C" void cleanupcuda() {
//	hipFree(dev_volume);
//	hipFreeArray(d_transferFuncArray);
//}

// Helper function for using CUDA to add vectors in parallel.
extern "C" void addWithCuda(std::vector<int>  &c, const std::vector<int> a, const std::vector<int> b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a.data(), size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b.data(), size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size >>> (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c.data(), dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	//return cudaStatus;
}

extern "C"
void freeCudaBuffers()
{
	checkCudaErrors(hipFreeArray(d_volumeArray));
	checkCudaErrors(hipFreeArray(d_transferFuncArray));
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, unsigned int *d_output, unsigned int imageW, unsigned int imageH,
	float density, float transferOffset, float3 dim, float3 ratio)
{
	d_render <<<gridSize, blockSize >>>(d_output, imageW, imageH, density,  transferOffset, dim, ratio);
}

extern "C" void copyInvViewMatrix(std::vector<float> pInvViewMatrix)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), pInvViewMatrix.data(), sizeof(float)*pInvViewMatrix.size()));
}




