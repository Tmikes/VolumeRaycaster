#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "float.h"
//#include "hip/hip_runtime_api.h"
#include "volumehelper.h"
//#include "volumehelper.cuh"

#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int  uint;
typedef unsigned char uchar;
//hipArray *d_volumeArray = 0;
hipArray *d_volumeArray;
hipArray *d_transferFuncArray;
int size = 0;
int3 dim = { 0,0,0 };


texture<uchar, hipTextureType3D, hipReadModeNormalizedFloat> volumeTex;
texture<float4, hipTextureType3D, hipReadModeElementType> transferTex;
surface<void, cudaSurfaceType3D> volumeSurf;
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

int iDivUp(int a, int b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}


#include "raycaster.cuh"
#include "logscale.cuh"
#include "blurdata.cuh"
#include "circlevertex.cuh"

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}






extern "C" void initCuda( std::vector<unsigned char> h_volume, int3 pDim)
{
	size = pDim.x *pDim.y*pDim.z;
	dim = pDim;
	
	//create 3D global texture
	hipChannelFormatDesc channelDescVolume = hipCreateChannelDesc<unsigned char>();
	hipExtent vol_dim = { pDim.x, pDim.y, pDim.z };
	hipMalloc3DArray(&d_volumeArray, &channelDescVolume, vol_dim, hipArraySurfaceLoadStore);
	// copy data to 3D array
	hipMemcpy3DParms copyParamsVol = { 0 };
	copyParamsVol.srcPtr = make_hipPitchedPtr(h_volume.data(), vol_dim.width * sizeof(unsigned char), vol_dim.width, vol_dim.height);
	copyParamsVol.dstArray = d_volumeArray;
	copyParamsVol.extent = vol_dim;
	copyParamsVol.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParamsVol);

	// set texture parameters
	volumeTex.normalized = true;                      // access with normalized texture coordinates
	volumeTex.filterMode = hipFilterModeLinear;      // linear interpolation
	volumeTex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	volumeTex.addressMode[1] = hipAddressModeClamp;
	//volumeTex.addressMode[2] = hipAddressModeClamp;

	// Bind the array to the texture
	hipBindTextureToArray(volumeTex, d_volumeArray, channelDescVolume);
	checkCudaErrors(cudaBindSurfaceToArray(volumeSurf, d_volumeArray));
	//---------------------transfer tex---------------------------------------------------------

	// create transfer function texture
	float4 transferFunc[] =
	{
		//----------		
			{ 1.0, 0.0, 0.0, 0.0, },
			{ 1.0, 0.0, 0.0, 1.0, },
			{ 1.0, 0.5, 0.0, 1.0, },
			{ 1.0, 1.0, 0.0, 1.0, },
			{ 0.0, 1.0, 0.0, 1.0, },
			{ 0.0, 1.0, 1.0, 1.0, },
			{ 0.0, 0.0, 1.0, 1.0, },
			{ 0.0, 0.0, 1.0, 1.0, },
			{ 1.0, 0.0, 0.0, 1.0, },
		//----------		
			{ 1.0, 0.0, 0.0, 0.0, },
			{ 1.0, 0.0, 0.0, 0.0, },
			{ 1.0, 0.5, 0.0, 0.0, },
			{ 1.0, 1.0, 0.0, 1.0, },
			{ 0.0, 1.0, 0.0, 1.0, },
			{ 0.0, 1.0, 1.0, 1.0, },
			{ 0.0, 0.0, 1.0, 1.0, },
			{ 0.0, 0.0, 1.0, 1.0, },
			{ 1.0, 0.0, 0.0, 1.0, },
		//----------		
			{ 1.0, 0.0, 0.0, 0.0, },
			{ 1.0, 0.0, 0.0, 0.0, },
			{ 1.0, 0.5, 0.0, 0.0, },
			{ 1.0, 1.0, 0.0, 0.0, },
			{ 0.0, 1.0, 0.0, 0.0, },
			{ 0.0, 1.0, 1.0, 0.5, },
			{ 0.0, 0.0, 1.0, 1.0, },
			{ 0.0, 0.0, 1.0, 1.0, },
			{ 1.0, 0.0, 0.0, 1.0, },
	};


	//hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
	//checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc) / sizeof(float4), 1));
	//checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

	//transferTex.filterMode = hipFilterModeLinear;
	//transferTex.normalized = true;    // access with normalized texture coordinates
	//transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

	//													 // Bind the array to the texture
	//checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
	hipExtent tf_dim = { 9, 3, 1 };
	hipMalloc3DArray(&d_transferFuncArray, &channelDesc, tf_dim);
	// copy data to 3D array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(transferFunc, tf_dim.width * sizeof(float4), tf_dim.width, tf_dim.height);
	copyParams.dstArray = d_transferFuncArray;
	copyParams.extent = tf_dim;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	
	// set texture parameters
	transferTex.normalized = true;                      // access with normalized texture coordinates
	transferTex.filterMode = hipFilterModeLinear;      // linear interpolation
	transferTex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	transferTex.addressMode[1] = hipAddressModeClamp;

											  //Bind the array to the texture
	hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc);
}


// Helper function for using CUDA to add vectors in parallel.
extern "C" void addWithCuda(std::vector<int>  &c, const std::vector<int> a, const std::vector<int> b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a.data(), size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b.data(), size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size >>> (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c.data(), dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	//return cudaStatus;
}

extern "C"
void freeCudaBuffers()
{
	checkCudaErrors(hipFreeArray(d_volumeArray));
	checkCudaErrors(hipFreeArray(d_transferFuncArray));
}


extern "C" void copyInvViewMatrix(std::vector<float> pInvViewMatrix)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), pInvViewMatrix.data(), sizeof(float)*pInvViewMatrix.size()));
}




